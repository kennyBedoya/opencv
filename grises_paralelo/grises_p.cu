#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>

using namespace cv;

//kernel
__global__ void image_gray(unsigned char *image_in, int width, int height, unsigned char *image_out){
int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;

if((row < height) && (col < width)){
  image_out[row*width+col] = image_in[(row*width+col)*3+ 2]*0.299 + image_in[(row*width+col)*3+ 1]*0.587 + image_in[(row*width+col)*3]*0.114;
  }


}

int main( int argc, char** argv ){

 //imagen de entrada
 char* imageName = argv[1];

 Mat image;
 image = imread( imageName, 1 );

 if( argc != 2 || !image.data )
 {
   printf( " No image data \n " );
   return -1;
 }

 //tamano imagen recibida
 Size tam = image.size();

 int width = tam.width;
 int height = tam.height;

 //reservar memoria
 unsigned char *data_image, *d_data_image,*image_output, *d_image_output;

 int tam_i = sizeof(unsigned char)*width*height;
 int tam_gray = sizeof(unsigned char)*width*height;

 data_image = (unsigned char*)malloc(tam_i);
 hipMalloc((void**)&d_data_image,tam_i);

 image_output = (unsigned char*)malloc(tam_gray);
 hipMalloc((void**)&d_image_output,tam_gray);


data_image = image.data;

//PRUEBA
 Mat prueba;
 prueba.create(height, width, CV_8UC1);
 prueba.data = image.data;

 namedWindow( "image PRUEBA", CV_WINDOW_AUTOSIZE);
 imshow( "image PRUEBA", prueba );
//FIN PRUEBA


//pasando al device
hipMemcpy(d_data_image,data_image,tam_i, hipMemcpyHostToDevice);

 int blockSize = 32;
 dim3 dimBlock(blockSize,blockSize,1);
 dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
 image_gray<<<dimGrid,dimBlock>>>(d_data_image,width,height,d_image_output);
 hipDeviceSynchronize();

 //pasando al host
 hipMemcpy(image_output,d_image_output,tam_gray,hipMemcpyDeviceToHost);

 //creando imagenes
 Mat gray_image;
 gray_image.create(height, width, CV_8UC1);
 gray_image.data = image_output;


 //mostrar imagenes
 imwrite( "../../images/Gray_Image.jpg", gray_image );

 namedWindow( imageName, CV_WINDOW_AUTOSIZE); //CV_WINDOW_AUTOSIZE
 namedWindow( "Gray image CUDA", CV_WINDOW_AUTOSIZE);

 imshow( imageName, image );
 imshow( "Gray image CUDA", gray_image );

 waitKey(0);

 hipFree(d_data_image);
 hipFree(d_image_output);

 return 0;

}
