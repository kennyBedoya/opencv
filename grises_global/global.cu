#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH 3

using namespace cv;

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

__global__ void sobelFilter_global(unsigned char *imageInput, int width, int height, unsigned int maskWidth,\
        char *M,unsigned char *imageOutput){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0;

    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    for(int i = 0; i < maskWidth; i++){
        for(int j = 0; j < maskWidth; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < width) \
                    &&(N_start_point_row + i >=0 && N_start_point_row + i < height)){
                Pvalue += imageInput[(N_start_point_row + i)*width+(N_start_point_col + j)] * M[i*maskWidth+j];
            }
        }
    }
    imageOutput[row*width+col] = clamp(Pvalue);
}

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 \
                                     + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}

//---------------Función secuencial------------------------
unsigned char *imagenGrises(unsigned char *gray, int height, int width, Mat image){ 
    unsigned char *imagen_salida;
    for(int i=0; i<height; i++){
        for(int j=0; j<width; j++){
        gray[(i*width+j)]= 0.299*image.data[(i*width+j)*3+2] + 0.587*image.data[(i*width+j)*3+1] + 0.114*image.data[(i*width+j)*3];
    }    
    }
    imagen_salida=gray;
    return imagen_salida;
}

int main(int argc, char **argv){
    hipError_t error = hipSuccess;
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used;
    char h_M[] = {-1,0,1,-2,0,2,-1,0,1}, *d_M;
    char* imageName = argv[1];
    unsigned char *image_aux,*gray;
    unsigned char *h_ImageInput, *d_ImageInput, *d_imageOutput, *h_imageOutput, *d_sobelOutput;
    Mat image;
    image = imread(imageName, 1);

    if(argc !=2 || !image.data){
        printf("No image Data \n");
        return -1;
    }

    Size s = image.size();

    int width = s.width;
    int height = s.height;
    int size = sizeof(unsigned char)*width*height*image.channels();
    int sizeGray = sizeof(unsigned char)*width*height;
    
    //------------Imagenes en escala de grises secuencial----
    Mat image_gray;

    gray = (unsigned char*)malloc(size);
    start = clock();
    for(int i=0; i<height; i++){
        for(int j=0; j<width; j++){
        gray[(i*width+j)]= 0.299*image.data[(i*width+j)*3+2] + 0.587*image.data[(i*width+j)*3+1] + 0.114*image.data[(i*width+j)*3];
    }    
    }
    end=clock();
    image_gray.create(height,width,CV_8UC1);
    image_gray.data=gray;
    //----------------------------------------------

    h_ImageInput = (unsigned char*)malloc(size);
    error = hipMalloc((void**)&d_ImageInput,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_ImageInput\n");
        exit(-1);
    }

    error = hipMalloc((void**)&d_M,sizeof(char)*9);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_M\n");
        exit(-1);
    }

    h_imageOutput = (unsigned char *)malloc(sizeGray);
    error = hipMalloc((void**)&d_imageOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageOutput\n");
        exit(-1);
    }

    error = hipMalloc((void**)&d_sobelOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_sobelOutput\n");
        exit(-1);
    }

    h_ImageInput = image.data;

    startGPU = clock();

    error = hipMemcpy(d_ImageInput,h_ImageInput,size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de h_ImageInput a d_ImageInput \n");
        exit(-1);
    }

		error = hipMemcpy(d_M,h_M,sizeof(char)*9,hipMemcpyHostToDevice);
		if(error != hipSuccess){
				printf("Error copiando los datos de h_ImageInput a d_ImageInput \n");
				exit(-1);
		}

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    img2gray<<<dimGrid,dimBlock>>>(d_ImageInput,width,height,d_imageOutput);
    hipDeviceSynchronize();
    sobelFilter_global<<<dimGrid,dimBlock>>>(d_imageOutput,width,height,3,d_M,d_sobelOutput);
    hipMemcpy(h_imageOutput,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);
    endGPU = clock();

    Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageOutput;

    start = clock();
    Mat gray_image_opencv, grad_x, abs_grad_x;
    cvtColor(image, gray_image_opencv, CV_BGR2GRAY);
    Sobel(gray_image_opencv,grad_x,CV_8UC1,1,0,3,1,0,BORDER_DEFAULT);
    convertScaleAbs(grad_x, abs_grad_x);
    end = clock();


    imwrite("./Sobel_Image.jpg",gray_image);

    //namedWindow(imageName, WINDOW_NORMAL);
    //namedWindow("Gray Image CUDA secuencial", WINDOW_NORMAL);
    //namedWindow("Gray Image SOBEL global", WINDOW_NORMAL);
    //namedWindow("Sobel Image OpenCV", WINDOW_NORMAL);

    //imshow(imageName,image);
    //imshow("Gray Image CUDA secuencial", image_gray);
    //imshow("Gray Image SOBEL global", gray_image);
    //imshow("Sobel Image OpenCV",abs_grad_x);
    waitKey(0);

    //free(h_ImageInput);    
    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo sobel global: %.10f\n",gpu_time_used);
    cpu_time_used = ((double) (end - start)) /CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo secuencial: %.10f\n",cpu_time_used);
    printf("La aceleración obtenida es de %.10fX\n",cpu_time_used/gpu_time_used);

    hipFree(d_ImageInput);
    hipFree(d_imageOutput);
    hipFree(d_M);
    hipFree(d_sobelOutput);
    return 0;
}
